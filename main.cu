// Ivlev Andrey B19 - 511
// Variant generation:
const int ID = 4; // Student ID
const int G = 511; // Group
const int X = G * 2 + ID; // X = 1026
const int A = X % 4; // A = 2
const int B = 5 + X % 5; // B = 6


#include <hip/hip_runtime.h>
#include <iostream>

__device__ int isSuitable(int R, int G, int B)
{
    if (R * G * B < 1000) return 1;
    else return 0;
}

__global__ void _cuda_parallel_pixels_counting(int size, int* data, int* d_number_of_pixels)
{
    int globalIdx = blockIdx.x * blockDim.x + threadIdx.x;
    for (int i = globalIdx; i < size; i+=3*B)
    {
        atomicAdd(&d_number_of_pixels[blockIdx.x], isSuitable((int)data[i], (int)data[i + 1], (int)data[i + 2]));
        // globalIdx += blockDim.x * gridDim.x;
        __syncthreads();
    }
}

int main() {
    // BMP file reading
    const char* filename = (char*) R"(C:\Users\warcr\CLionProjects\ParallelProgrammingCUDA\images\img01.bmp)";

    int i;
    FILE* f = fopen(filename, "rb");
    unsigned char info[54];

    // read the 54-byte header
    fread(info, sizeof(unsigned char), 54, f);

    // extract image height and width from header
    int width = *(int*)&info[18];
    int height = *(int*)&info[22];
    std::cout << "width: " << width << std::endl;
    std::cout << "height: " << height << std::endl;

    // allocate 3 bytes per pixel
    int size = 3 * width * height;
    auto* data = new unsigned char[size];

    // read the rest of the data at once
    fread(data, sizeof(unsigned char), size, f);
    fclose(f);
    for (i = 0; i < size; i += 3)
    {
        // flip the order of every 3 bytes in order to get RGB instead of BGR
        unsigned char tmp = data[i];
        data[i] = data[i + 2];
        data[i + 2] = tmp;
    }
    // data is read

    int* number_of_pixels = new int[B];
    int* d_data;
    int* d_number_of_pixels;
    hipMalloc(&d_data, size * sizeof(int));
    hipMalloc(&d_number_of_pixels, B * sizeof(int));
    hipMemcpy(d_data, data, size * sizeof(int), hipMemcpyHostToDevice);

    dim3 gridSize = dim3(1, 1, 1);
    dim3 blockSize = dim3(B, 1, 1);

    _cuda_parallel_pixels_counting<<<gridSize, blockSize>>>(size, d_data, d_number_of_pixels);

    hipDeviceSynchronize();
    hipMemcpy(number_of_pixels, d_number_of_pixels, B * sizeof(int), hipMemcpyDeviceToHost);
    hipFree(d_data);
    hipFree(d_number_of_pixels);

    int total_number_of_pixels = 0;
    std::cout << "Thread results: [";
    for (int i = 0; i < B; i++)
    {
        total_number_of_pixels += number_of_pixels[i];
        std::cout << number_of_pixels[i];
        if (i != B-1)  std::cout << " ";
    }
    std::cout << "]" << std::endl;

    std::cout << "Total number of suitable pixels (CUDA): " << total_number_of_pixels << std::endl;
    std::cout << std::endl;
    return 0;
}
